#include <stdio.h>
#include <stdlib.h>
#include<algorithm>
#include <hip/hip_runtime.h>
using namespace std;

//INSERT CODE HERE---------------------------------
__global__ void make_count_table(int* src, int* count,int size){
	__shared__ int dataShared[101];
	for(int i =0;i<101;i++){
		dataShared[i]=0;
	}
	int gx = blockIdx.x * blockDim.x + threadIdx.x,
		 tx = threadIdx.x;
	if(gx<size){
		atomicAdd(&dataShared[src[gx]],1);
	}
	__syncthreads();
	if(tx<101)
		atomicAdd(&count[tx],dataShared[tx]);
}
__global__ void make_offset_table(int* count){
	__shared__ int src[101],dst[101];
	int tx = threadIdx.x,
	    temp;
	src[tx] = count[tx];
	for(int stride=1;stride<101;stride*=2){
		__syncthreads();
		if(tx-stride>=0)
			dst[tx] = src[tx] + src[tx-stride];
		else
			dst[tx] = src[tx];
		temp=dst[tx];
                dst[tx]=src[tx];
                src[tx]=temp;
	}
	__syncthreads();
	count[tx] = src[tx];

}
__global__ void sort(int* src,int* dst,int* offset,int size){
	//__shared__ int OffsetShared[101];
	int gx = blockIdx.x * blockDim.x + threadIdx.x;
	int n;
	/*if(tx<101)
		OffsetShared[tx]=offset[tx];
	__syncthreads();
*/
	if(gx<size){
		n=atomicSub(&offset[src[gx]],1);
		n=n-1;
		//printf("dst[%d] = src[%d]\n",n,gx);
		dst[n]=src[gx];
	}
}

void verify(int* src, int*result, int input_size){
	sort(src, src+input_size);
	long long match_cnt=0;
	for(int i=0; i<input_size;i++)
	{
		if(src[i]==result[i])
			match_cnt++;
	}

	if(match_cnt==input_size)
		printf("TEST PASSED\n\n");
	else
		printf("TEST FAILED\n\n");

}

void genData(int* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (int)(rand() % 101);
	}
}

int main(int argc, char* argv[]) {
	int* pSource = NULL;
	int* pResult = NULL;
	int* pCount = NULL;
	int input_size=0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (argc == 2)
		input_size=atoi(argv[1]);
	else
	{
    		printf("\n    Invalid input parameters!"
	   		"\n    Usage: ./sort <input_size>"
           		"\n");
        	exit(0);
	}

	//allocate host memory
	pSource=(int*)malloc(input_size*sizeof(int));
	pResult=(int*)malloc(input_size*sizeof(int));
	pCount=(int*)malloc(101*sizeof(int));
	// generate source data
	genData(pSource, input_size);
/*	for(int i=0;i<input_size;i++)
		printf("pSource[%d] = %d\n",i,pSource[i]);*/
	//allocate device memory
	int* pSourceDev = NULL;
	int* pResultDev = NULL;
	int* pCountDev = NULL;
	hipMalloc((void**)&pSourceDev,input_size*sizeof(int));
	hipMalloc((void**)&pResultDev,input_size*sizeof(int));
	hipMalloc((void**)&pCountDev,101*sizeof(int));	
	// start timer
	hipEventRecord(start, 0);
	hipMemcpy(pSourceDev, pSource, input_size * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(ceil((float)input_size/(float)1024),1,1);
	dim3 dimBlock(1024,1,1);
	make_count_table<<<dimGrid,dimBlock>>>(pSourceDev,pCountDev,input_size);
	hipMemcpy(pCount,pCountDev,101*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int i=0;i<101;i++){
                printf("Count[%d] = %d\n",i,pCount[i]);
        }*/


	dim3 dimGrid2(1,1,1);
	dim3 dimBlock2(101,1,1);
	make_offset_table<<<dimGrid2,dimBlock2>>>(pCountDev);
	hipMemcpy(pCount,pCountDev,101*sizeof(int),hipMemcpyDeviceToHost);
	/*for(int i=0;i<101;i++){
		printf("Offset[%d] = %d\n",i,pCount[i]);
	}*/
	//cudaMemcpy(pResult, pResultDev, input_size * sizeof(int), cudaMemcpyDeviceToHost);
	dim3 dimGrid3(ceil((float)input_size/(float)1024),1,1);
        dim3 dimBlock3(1024,1,1);
	sort<<<dimGrid3,dimBlock3>>>(pSourceDev,pResultDev,pCountDev,input_size);
	hipMemcpy(pResult, pResultDev, input_size * sizeof(int), hipMemcpyDeviceToHost);
	/*for(int i=0;i<input_size;i++)
		printf("Reslut[%d]= %d\n",i,pResult[i]);
*/
	//INSERT CODE HERE--------------------
	





	// end timer
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("elapsed time = %f msec\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
    	printf("Verifying results..."); fflush(stdout);
	verify(pSource, pResult, input_size);
	fflush(stdout);
	/*for( int i=0;i<input_size;i++){
                printf("index %d : %d %d \n",i,pSource[i],pResult[i]);
        }*/

}


